#include <stdio.h>
#include <nvToolsExt.h>
#include "kernel.cuh"


int main() {
    const int N = 1024;
    float *ha, *hb, *hc;
    float *da, *db, *dc;

    // Allocate Host memory
    // malloc returns a pointer to start of allocated space
    // Cast the pointer to be of type ha
    ha = (float*) malloc(N * sizeof(float));
    hb = (float*) malloc(N * sizeof(float));
    hc = (float*) malloc(N * sizeof(float));

    // Initialize these arrays
    for (int i=0; i<N; ++i){
        ha[i] = i;
        hb[i] = 2 * i;
    }

    // Allocate Device memory
    // Why are we giving address of a pointer to a pointer?
    hipMalloc(&da, N * sizeof(float));
    hipMalloc(&db, N * sizeof(float));
    hipMalloc(&dc, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(da, ha, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, hb, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch Kernel
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    vectorAdd<<<numBlocks, blockSize>>>(da, db, dc, N);

    // Move result back to host
    hipMemcpy(hc, dc, N * sizeof(float), hipMemcpyDeviceToHost);

    // Verify
    for(int i=0; i < N; ++i){
        if(hc[i] != ha[i] + hb[i]){
            printf("Verification failed at element %d\n", i);
            break;
        }
    }
    printf("Done");


    // Cleanup
    hipFree(da); hipFree(db); hipFree(dc);
    free(ha); free(hb); free(hc);

    return 0;
}